#include "hip/hip_runtime.h"
#include <cstdio>
#include <math.h>
#include <iostream>
#include <fstream>
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

using namespace std;

__global__ void cal(double *u, double *v, double *p, double *b, double *un, double *vn, double *pn, int nx, int ny, int nt, int nit, double dx, double dy, double dt, int rho, double nu) {
        int j = blockIdx.x;
	int i = threadIdx.x;
        int jp = (j+1)*ny+i;
	int jm = (j-1)*ny+i;
	int ip = j*ny+(i+1);
	int im = j*ny +(i-1);
	int ji = j*ny +i;
	grid_group grid = this_grid();	
//        for (int n = 0; n < nt; n++) {
		
		if (j != 0 && j != ny-1 && i != 0 && i != nx-1) {
        		b[ji] = rho*((1/dt) * ((u[ip] - u[im]) / (2*dx) + (v[jp] - v[jm]) / (2*dy)) - pow(((u[ip]-u[im]) /\
			(2*dx)), 2) - 2*(((u[jp] - u[jm]) / (2*dy)) * (v[ip] - v[im]) / (2*dx)) - pow(((v[jp] - v[jm]) / (2*dy)),2));
        	}
		grid.sync();
		
	        for (int it = 0; it < nit; it++) {
                	pn[ji] = p[ji];
                        
                        if (j != 0 && j != ny-1 && i != 0 && i != nx-1) {     
                                
                                p[ji] = (pow(dy,2) * (pn[ip] + pn[im]) + pow(dx,2) * (pn[jp] + pn[jm]) - b[ji] * pow(dx,2) * pow(dy,2)) / (2*(pow(dx,2) + pow(dy,2)));
                        }
                        
			grid.sync();
                        
                       p[(j*ny)+(ny-1)] = p[(j*ny)+(ny-2)];
                       p[i] = p[ny+i];
                       p[j*ny] = p[(j*ny)+1];
                       p[(ny * (nx-1))+i] = 0;
                       grid.sync(); 

                }
		un[ji] = u[ji];
                vn[ji] = v[ji];
		grid.sync();
                        if (j != 0 && j != ny-1 && i != 0 && i != nx-1) {

                         
                        
                                u[ji] = un[ji] - un[ji] * dt / dx * (un[ji] - un[im]) - vn[ji] * dt / dy * (un[ji] - un[jm]) - dt / (2*rho*dx) *\
                                (p[ip] - p[im]) + nu * dt / pow(dx,2) * (un[ip] - 2 * un[ji] + un[im]) + nu * dt / pow(dy,2) *\
                                (un[jp] - 2 * un[ji] + un[jm]);
                                v[ji] = vn[ji] - un[ji] * dt / dx * (vn[ji] - vn[im]) - vn[ji] * dt / dy * (vn[ji] - vn[jm]) - dt / (2*rho*dx) *\
                                (p[jp] - p[jm]) + nu * dt / pow(dx,2) * (vn[ip] - 2 * vn[ji] + vn[im]) + nu * dt / pow(dy,2) *\
                                (vn[jp] - 2 * vn[jp] + vn[jm]);

                        }

		grid.sync();
                
		
                u[i] = u[j*ny] = u[j*ny +(ny-1)] = 0.0;
                u[ny*(nx-1) + i] = 1.0;
                v[i] = v[ny*(nx-1)+i] = v[j*ny] = v[j*ny + (ny-1)] = 0.0;
                
//		if (blockIdx.x == 39) printf("%f ", v[ji]);

 		
//	}
                
}


int main() {
	const int nx = 41;
       	const int ny = 41;
	const int nt = 80;
	const int nit = 50;
	const double dx = 2.0 / (nx - 1);
	const double dy = 2.0 / (ny-1);
	const double dt = 0.01;
	const int rho = 1;
	const double nu = 0.02;
	double* u; hipMallocManaged(&u, nx*ny*sizeof(double));
        double* v; hipMallocManaged(&v, nx*ny*sizeof(double));
        double* p; hipMallocManaged(&p, nx*ny*sizeof(double));
        double* b; hipMallocManaged(&b, nx*ny*sizeof(double));
        double* un; hipMallocManaged(&un, nx*ny*sizeof(double));
        double* vn; hipMallocManaged(&vn, nx*ny*sizeof(double));
        double* pn; hipMallocManaged(&pn, nx*ny*sizeof(double));
/*
	u[nx*ny] = {0};
	v[nx*ny] = {0};
	p[nx*ny] = {0};
	b[nx*ny] = {0};
	pn[nx*ny] = {0};
	un[nx*ny] = {0};
	vn[nx*ny] = {0}; 
*/
	hipMemset(u, 0, nx*ny*sizeof(double));
        hipMemset(v, 0, nx*ny*sizeof(double));
        hipMemset(p, 0, nx*ny*sizeof(double));
        hipMemset(b, 0, nx*ny*sizeof(double));
        hipMemset(un, 0, nx*ny*sizeof(double));
        hipMemset(vn, 0, nx*ny*sizeof(double));
        hipMemset(pn, 0, nx*ny*sizeof(double));

	ofstream ufile("u.dat");
	ofstream vfile("v.dat");
	ofstream pfile("p.dat");


	for (int n = 0; n < nt; n++) {
//		cal<<<nx,ny>>>(u, v, p, b, un, vn, pn, nx, ny, nt, nit, dx, dy, dt, rho, nu);
                void *args[] = {(void *)&u, (void *)&v, (void *)&p, (void *)&b, (void *)&un, (void *)&vn, (void *)&pn, (void *)&nx, (void *)&ny, (void *)&n, (void *)&nit, \
               (void *)&dx, (void *)&dy, (void *)&dt, (void *)&rho, (void *)&nu };
                hipLaunchCooperativeKernel((void*)cal, nx, ny, args);
		
 		hipDeviceSynchronize();
	
		
        if (n % 10 == 0) {
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          ufile << u[j*ny+i] << " ";
      ufile << "\n";
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          vfile << v[(j*ny)+i] << " ";
      vfile << "\n";
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          pfile << p[j*ny+i] << " ";
      pfile << "\n";
                   }

        

	}
	hipFree(u);
	hipFree(v);
	hipFree(p);
	hipFree(b);
	hipFree(un);
	hipFree(vn);
	hipFree(pn); 
  ufile.close();
  vfile.close();
  pfile.close();



}

