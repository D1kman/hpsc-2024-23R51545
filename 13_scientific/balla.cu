
#include <hip/hip_runtime.h>
#include <cstdio>
#include <math.h>
#include <iostream>
#include <fstream>
using namespace std;

__global__ void cal(double *u, double *v, double *p, double *b, double *un, double *vn, double *pn, int nx, int ny, int n, int nit, double dx, double dy, double dt, int rho, double nu) {
        int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	printf("%d ", j);
       // for (int n = 0; n < nt; n++) {
	
		if (j % ny != 0 && j % ny != 40) {
        		b[j] = rho*((1/dt) * ((u[j+1] - u[j-1]) / (2*dx) + (v[j+nx] - v[j-nx]) / (2*dy)) - pow(((u[j+1]-u[j-1]) / (2*dx)), 2) -\
                	2*(((u[j+nx] - u[j-nx]) / (2*dy)) * (v[j+1] - v[j-1]) / (2*dx)) - pow(((v[j+nx] - v[j-nx]) / (2*dy)),2));
        	}
		__syncthreads();
	        for (int it = 0; it < nit; it++) {
                	pn[j] = p[j];
                        
                        if (j % ny != 0 && j % ny != 40) {     
                                
                                p[j] = (pow(dy,2) * (pn[j+1] + pn[j-1]) + pow(dx,2) * (pn[j+nx] + pn[j-nx]) - b[j] * pow(dx,2) * pow(dy,2)) / (2*(pow(dx,2) + pow(dy,2)));
                        }
                        
			__syncthreads();
                        if (j < nx) {
                                p[(j*nx)+(ny-1)] = p[(j*nx)+(ny-2)];
                                p[j] = p[ny+j];
                                p[j * ny] = p[(j*ny)+1];
                                p[(ny * (nx-1))+j] = 0;
                        }

                }
		un[j] = u[j];
                vn[j] = v[j];
		__syncthreads();
                        if (j % ny != 0 && j % ny != 40) {

                         
                        
                                u[j] = un[j] - un[j] * dt / dx * (un[j] - un[j-1]) - vn[j] * dt / dy * (un[j] - un[j-ny]) - dt / (2*rho*dx) *\
                                (p[j+1] - p[j-1]) + nu * dt / pow(dx,2) * (un[j+1] - 2 * un[j] + un[j-1]) + nu * dt / pow(dy,2) *\
                                (un[j+ny] - 2 * un[j] + un[j-ny]);
                                v[j] = vn[j] - un[j] * dt / dx * (vn[j] - vn[j-1]) - vn[j] * dt / dy * (vn[j] - vn[j-ny]) - dt / (2*rho*dx) *\
                                (p[j+ny] - p[j-ny]) + nu * dt / pow(dx,2) * (vn[j+1] - 2 * vn[j] + vn[j-1]) + nu * dt / pow(dy,2) *\
                                (vn[j+ny] - 2 * vn[j] + vn[j-ny]);

                        }
		__syncthreads();
                if (j < nx) {
                        u[j] = u[j * ny] = u[j * ny + (ny-1)] = 0.0;
                        u[ny*(nx-1) + j] = 1.0;
                        v[j] = v[ny*(nx-1)+j] = v[j*ny] = v[j*ny + (ny-1)] = 0.0;
                }
 		__syncthreads();


//	}
                
        

}


int main() {
	const int nx = 41;
       	const int ny = 41;
	const int nt = 50;
	const int nit = 50;
	const double dx = 2.0 / (nx - 1);
	const double dy = 2.0 / (ny-1);
	const double dt = 0.01;
	const int rho = 1;
	const double nu = 0.02;
	double* u; hipMallocManaged(&u, nx*ny*sizeof(double));
        double* v; hipMallocManaged(&v, nx*ny*sizeof(double));
        double* p; hipMallocManaged(&p, nx*ny*sizeof(double));
        double* b; hipMallocManaged(&b, nx*ny*sizeof(double));
        double* un; hipMallocManaged(&un, nx*ny*sizeof(double));
        double* vn; hipMallocManaged(&vn, nx*ny*sizeof(double));
        double* pn; hipMallocManaged(&pn, nx*ny*sizeof(double));
        const int M = 1024;
        const int N = (nx-1)*(ny-1);
	u[nx*ny] = {0};
	v[nx*ny] = {0};
	p[nx*ny] = {0};
	b[nx*ny] = {0};
	pn[nx*ny] = {0};
	un[nx*ny] = {0};
	vn[nx*ny] = {0};

	  ofstream ufile("u.dat");
	  ofstream vfile("v.dat");
	  ofstream pfile("p.dat");


	for (int n = 0; n < nt; n++) {
		printf("walla");
		cal<<<(N+M-1)/M,M>>>(u, v, p, b, un, vn, pn, nx, ny, n, nit, dx, dy, dt, rho, nu);
/*
		for (int j = ny; j < (nx-1)*(ny); j++){
			if (j % ny == 0 || j % ny == 40) {
				
			}
			else {
				b[j] = rho*((1/dt) * ((u[j+1] - u[j-1]) / (2*dx) + (v[j+nx] - v[j-nx]) / (2*dy)) - pow(((u[j+1]-u[j-1]) / (2*dx)), 2) -\
				2*(((u[j+nx] - u[j-nx]) / (2*dy)) * (v[j+1] - v[j-1]) / (2*dx)) - pow(((v[j+nx] - v[j-nx]) / (2*dy)),2));
			}
		}
*/
/*
		for (int it = 0; it < nit; it++) {
			for(int j = 0; j < (nx)*(ny); j++) {
				pn[j] = p[j];
			}

			for(int j = ny; j < (nx-1)*(ny); j++){
				if (j % ny == 0 || j % ny == 40) {
					
				}
				else {
				p[j] = (pow(dy,2) * (pn[j+1] + pn[j-1]) + pow(dx,2) * (pn[j+nx] + pn[j-nx]) - b[j] * pow(dx,2) * pow(dy,2)) / (2*(pow(dx,2) + pow(dy,2)));
				}	
			}

			for (int row = 0; row < nx; row++) {
				p[(row*nx)+(ny-1)] = p[(row*nx)+(ny-2)];
				p[row] = p[ny+row];
				p[row * ny] = p[(row*ny)+1];
				p[(ny * (nx-1))+row] = 0;
			}
		
		}
*/
/*		
		for (int row = 0; row < (nx)*(ny); row++) {
			un[row] = u[row];
			vn[row] = v[row];	
		}
*/
/*		
		for (int j = ny; j < (nx-1)*(ny); j++) {
			if (j % ny == 0 || j % ny == 40) {

                         }
                        else {
				u[j] = un[j] - un[j] * dt / dx * (un[j] - un[j-1]) - vn[j] * dt / dy * (un[j] - un[j-ny]) - dt / (2*rho*dx) *\
				(p[j+1] - p[j-1]) + nu * dt / pow(dx,2) * (un[j+1] - 2 * un[j] + un[j-1]) + nu * dt / pow(dy,2) *\
				(un[j+ny] - 2 * un[j] + un[j-ny]);
				v[j] = vn[j] - un[j] * dt / dx * (vn[j] - vn[j-1]) - vn[j] * dt / dy * (vn[j] - vn[j-ny]) - dt / (2*rho*dx) *\
                                (p[j+ny] - p[j-ny]) + nu * dt / pow(dx,2) * (vn[j+1] - 2 * vn[j] + vn[j-1]) + nu * dt / pow(dy,2) *\
                                (vn[j+ny] - 2 * vn[j] + vn[j-ny]);

			}
		}
*/
/*		
		for(int row = 0; row < nx; row++) {
			u[row] = u[row * ny] = u[row * ny + (ny-1)] = 0.0;
			u[ny*(nx-1) + row] = 1.0;
			v[row] = v[ny*(nx-1)+row] = v[row*ny] = v[row*ny + (ny-1)] = 0.0;
		}
*/	
		
	if (n % 10 == 0) {
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          ufile << u[j*ny+i] << " ";
      ufile << "\n";
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          vfile << v[(j*ny)+i] << " ";
      vfile << "\n";
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          pfile << p[j*ny+i] << " ";
      pfile << "\n";
		   }
 	hipDeviceSynchronize();
	hipFree(u);
	hipFree(v);
	hipFree(p);
	hipFree(b);
	hipFree(un);
	hipFree(vn);
	hipFree(pn);  
	}
  ufile.close();
  vfile.close();
  pfile.close();



}

