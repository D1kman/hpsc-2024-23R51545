
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void bucket_sort(int *bucket, int *key,  int range, int n) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id < range) {
  bucket[id] = 0;
  }
  __syncthreads();

  atomicAdd(&bucket[key[id]],1);
  __syncthreads();
  int offset = 0;
  for (int i = 0; i < range; i++) {
	if (id  < bucket[i]+offset && id >= offset) {
	  key[id] = i;
	}
	  offset += bucket[i];  
  }

}

int main() {
  int n = 50;
  int range = 5;
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  int *bucket;
  hipMallocManaged(&bucket, range*sizeof(int));
 
  
  bucket_sort<<<1, n>>>(bucket, key, range, n);
  
  hipDeviceSynchronize();
  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");

  hipFree(bucket);
}
